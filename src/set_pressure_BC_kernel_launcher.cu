// !**************************************************************************************************                                    // !* by Hadi Zolfaghari, ARTORG Center, University of Bern, (zolfaghari.haadi@gmail.com)            *      
// !* October 2015 - March 2020                                                                      *            
// !* Modified by Hadi Zolfaghari, University of Cambridge (hz382@damtp.cam.ac.uk)                   *      
// !* April 2020 -                                                                                   *            
// !**************************************************************************************************
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "set_pressure_BC_kernel.cu"
extern "C" void pressure_bc_launcher_(int *type_id1, int *type_id2, int *type_id3, int *k_start_index, int *nx_dev, int *ny_dev, int *nz_dev, double *pre_inlet,  double *pre_outlet, double *pre_outlet_2, double** ct_geometry_dev, double** rel_dev)
{

/*dim3    blocks((nx_dev-8)/8, (ny_dev-8)/8, (nz_dev-8)/8);
*/

int nx = (*nx_dev-8)/8;
int ny = (*ny_dev-8)/8;
int nz = (*nz_dev-8)/8;


dim3 blocks(nx,ny,nz);

int nxt = (*nx_dev-8)/blocks.x;
int nyt = (*ny_dev-8)/blocks.y;
int nzt = (*nz_dev-8)/blocks.z;

/*printf("Value of n=%d", nxt);
printf("Value of n=%d", nx);*/

 
dim3 threads(nxt,nyt,nzt);

/*int nxg = *nx_dev;
int nyg = *ny_dev;
int nzg = *nz_dev; 

hipMemcpy(*phi_grad_dev, pre, sizeof(double) * nxg * nyg * nzg, hipMemcpyHostToDevice ); */

/*int nxgc1 = *nx_grad_coef1;
int nygc1 = *ny_grad_coef1;


hipMemcpy(*cGp1_dev, cGp1, sizeof(double) * nxgc1 * nygc1, hipMemcpyHostToDevice ); */


//int row = sizeof(*phi_grad_dev) / sizeof(*phi_grad_dev[0]);
//int column = sizeof(*phi_grad_dev[0])/row;
//printf ("pressure host pointer:, %f\n", pre[0]);
//printf ("pressure host:, %f\n", phi_grad_dev[0]);

/*    if (mod(nx_dev, szblock) .ne. 0) blocks%x = blocks%x + 1 */
/*dim3    threads((nx_dev-8)/blocks.x, (ny_dev-8)/blocks.y, (nz_dev-8)/blocks.z);
*/

set_pressure_BC_kernel<<< blocks,threads >>>(*type_id1, *type_id2, *type_id3, *k_start_index, *nx_dev, *ny_dev, *nz_dev, *pre_inlet, *pre_outlet, *pre_outlet_2, *ct_geometry_dev, *rel_dev);
return;
}
