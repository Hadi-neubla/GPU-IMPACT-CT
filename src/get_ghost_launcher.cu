// !**************************************************************************************************                                    // !* by Hadi Zolfaghari, ARTORG Center, University of Bern, (zolfaghari.haadi@gmail.com)            *      
// !* October 2015 - March 2020                                                                      *            
// !* Modified by Hadi Zolfaghari, University of Cambridge (hz382@damtp.cam.ac.uk)                   *      
// !* April 2020 -                                                                                   *            
// !**************************************************************************************************
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "get_ghost.cu"
extern "C" void get_ghost_launcher_(int *nx_dev, int *ny_dev, int *nz_dev, double** rel_dev, double** rear_ghost_dev, double** front_ghost_dev, double** upper_ghost_dev, double** lower_ghost_dev, double** west_ghost_dev, double** east_ghost_dev)
{

/*dim3    blocks((nx_dev-8)/8, (ny_dev-8)/8, (nz_dev-8)/8);
*/

int nx = (*nx_dev-8)/8;
int ny = (*ny_dev-8)/8;
int nz = (*nz_dev-8)/8;


dim3 blocks(nx,ny,nz);

int nxt = (*nx_dev-8)/blocks.x;
int nyt = (*ny_dev-8)/blocks.y;
int nzt = (*nz_dev-8)/blocks.z;

/*printf("Value of n=%d", nxt);
printf("Value of n=%d", nx);*/

 
dim3 threads(nxt,nyt,nzt);

/*int nxg = *nx_dev;
int nyg = *ny_dev;
int nzg = *nz_dev; 

hipMemcpy(*phi_grad_dev, pre, sizeof(double) * nxg * nyg * nzg, hipMemcpyHostToDevice ); */

/*int nxgc1 = *nx_grad_coef1;
int nygc1 = *ny_grad_coef1;


hipMemcpy(*cGp1_dev, cGp1, sizeof(double) * nxgc1 * nygc1, hipMemcpyHostToDevice ); */


//int row = sizeof(*phi_grad_dev) / sizeof(*phi_grad_dev[0]);
//int column = sizeof(*phi_grad_dev[0])/row;
//printf ("pressure host pointer:, %f\n", pre[0]);
//printf ("pressure host:, %f\n", phi_grad_dev[0]);

/*    if (mod(nx_dev, szblock) .ne. 0) blocks%x = blocks%x + 1 */
/*dim3    threads((nx_dev-8)/blocks.x, (ny_dev-8)/blocks.y, (nz_dev-8)/blocks.z);
*/

get_ghost<<< blocks,threads >>>(*nx_dev, *ny_dev, *nz_dev, *rel_dev, *rear_ghost_dev, *front_ghost_dev, *upper_ghost_dev, *lower_ghost_dev, *west_ghost_dev, *east_ghost_dev);
return;
}
