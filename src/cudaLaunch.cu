// !**************************************************************************************************                                           
// !* by Hadi Zolfaghari, ARTORG Center, University of Bern, (zolfaghari.haadi@gmail.com)            *      
// !* October 2015 - March 2020                                                                      *            
// !**************************************************************************************************
#include <dlfcn.h>
#include <stdio.h>

extern "C" int cudaLaunch_(const char* name)
{
	void* handle = dlopen(NULL, RTLD_NOW);
	if (!handle)
	{
		fprintf(stderr, "%s\n", dlerror());
		exit(-1);
	}
	void* func = dlsym(handle, name);
	if (!func)
	{
		fprintf(stderr, "%s\n", dlerror());
		exit(-2);
	}

	return hipLaunchByPtr(func);
}
